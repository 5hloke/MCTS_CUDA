#include "hip/hip_runtime.h"
#include "./../include/board.h"
__global__ void check_winner_kernel(Token *board, Token *winner, int size, int win_len)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > size || j > size)
    {
        return;
    }
    Token player = board[i * size + j];

    if (player == Token::EMPTY)
    {
        return;
    }
    const int n_len = win_len - 1;
    Token vertical_up[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token vertical_down[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token horizontal_left[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token horizontal_right[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token diag1[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token diag2[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token diag3[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    Token diag4[4] = {Token::EMPTY, Token::EMPTY, Token::EMPTY, Token::EMPTY};
    // Vertical Checks, horizontal checks, diagonals

    for (int k = 0; k < n_len; k++)
    {
        if (i + k < size)
        {
            vertical_up[k] = board[(i + k) * size + j];
        }
        if (i - k > 0)
        {
            vertical_down[k] = board[(i - k) * size + j];
        }
        if (j - k > 0)
        {
            horizontal_left[k] = board[i * size + (j - k)];
        }
        if (j + k < size)
        {
            horizontal_right[k] = board[i * size + (j + k)];
        }
        if (i + k < size && j + k < size)
        {
            diag1[k] = board[(i + k) * size + (j + k)];
        }
        if (i - k > 0 && j - k > 0)
        {
            diag2[k] = board[(i - k) * size + (j - k)];
        }
        if (i - k > 0 && j + k < size)
        {
            diag3[k] = board[(i - k) * size + (j + k)];
        }
        if (i + k < size && j - k > 0)
        {
            diag4[k] = board[(i + k) * size + (j - k)];
        }
    }
    // Check for winne
    int up = 1, down = 1, left = 1, right = 1, d1 = 1, d2 = 1, d3 = 1, d4 = 1;
    for (int k = 0; k < n_len; k++)
    {
        if (vertical_up[k] != player && up != 0)
        {
            up = 0;
        }
        if (vertical_down[k] != player && down != 0)
        {
            down = 0;
        }
        if (horizontal_left[k] != player && left != 0)
        {
            left = 0;
        }
        if (horizontal_right[k] != player && right != 0)
        {
            right = 0;
        }
        if (diag1[k] != player && d1 != 0)
        {
            d1 = 0;
        }
        if (diag2[k] != player && d2 != 0)
        {
            d2 = 0;
        }
        if (diag3[k] != player && d3 != 0)
        {
            d3 = 0;
        }
        if (diag4[k] != player && d4 != 0)
        {
            d4 = 0;
        }
    }
    if ((up == 1 || down == 1 || left == 1 || right == 1 || d1 == 1 || d2 == 1 || d3 == 1 || d4 == 1) && *winner == Token::EMPTY)
    {
        *winner = player;
        return;
    }
    else
    {
        return;
    }
}

__global__ void valid_moves_kernel(Token *device_board,
                                   int board_size,
                                   Position *valid_moves,
                                   int *valid_moves_count)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= board_size || j >= board_size)
    {
        return;
    }

    if (device_board[i * board_size + j] == Token::EMPTY)
    {
        // printf("I is %d and J is %d\n", i, j);
        int index = atomicAdd(valid_moves_count, 1);
        Position pos = {i, j};
        valid_moves[index] = pos;
    }
}

Board::Board()
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            m_board[i][j] = Token::EMPTY;
        }
    }
}

// Board::Board(const Board &other) : m_board(other.m_board) {} // this copy constructor is not correct

bool Board::valid_move(int row, int col) const
{
    return row >= 0 && row < BOARD_SIZE && col >= 0 && col < BOARD_SIZE && m_board[row][col] == Token::EMPTY;
}

void update_board(Board &other)
{
    for (int i = 0; i < other.m_board.size(); i++)
    {
        for (int j = 0; j < other.m_board[i].size(); j++)
        {
            m_board[i][j] = other.m_board[i][j];
        }
    }
}

bool Board::make_move(int row, int col, Token player)
{
    if (!valid_move(row, col))
    {
        return false;
    }
    m_board[row][col] = player;
    return true;
}


bool Board::has_winner() const
{
    return get_winner() != Token::EMPTY;
}

void Board::move_to_gpu()
{
    // This function is not required for the assignment
    Token *dummy = new Token[BOARD_SIZE * BOARD_SIZE];
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            dummy[i * BOARD_SIZE + j] = m_board[i][j];
        }
    }
    hipMalloc(&d_board, BOARD_SIZE * BOARD_SIZE * sizeof(Token)); // TODO :: Create a new variable for GPU . Do not use m_board. This has to be passed into the kernel
    hipMemcpy(d_board, dummy, BOARD_SIZE * BOARD_SIZE * sizeof(Token), hipMemcpyHostToDevice);
    delete[] dummy;
}
void Board::move_to_cpu()
{
    // This function is not required for the assignment
    Token *dummy = new Token[BOARD_SIZE * BOARD_SIZE];
    hipMemcpy(dummy, d_board, BOARD_SIZE * BOARD_SIZE * sizeof(Token), hipMemcpyDeviceToHost); // TODO :: Create a new variable for GPU . Do not use m_board. This has to be passed into the kernel
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; i < BOARD_SIZE; j++)
        {
            m_board[i][j] = dummy[i * BOARD_SIZE + j];
        }
    }
}
// CUDA kernel for get_winner needs to be written over here
/*void Board::clear_space()
{
    // This function is not required for the assignment
    hipFree(d_board);
}*/

// If theres no winner returns Token::EMPTY, if there is a winner return the player Token::BLACK/Token::WHITE
Token Board::get_winner() const
{
    Token winner = Token::EMPTY;
    Token dummy = Token::EMPTY;
    dim3 block(8, 8);
    dim3 grid(BOARD_SIZE / block.x + 1, BOARD_SIZE / block.y + 1);
    Token *d_winner = &dummy;
    hipMalloc(&d_winner, sizeof(Token));
    check_winner_kernel<<<grid, block>>>(d_board, d_winner, BOARD_SIZE, WINNING_LENGTH);
    hipMemcpy(&winner, d_winner, sizeof(Token), hipMemcpyDeviceToHost);
    hipFree(d_winner);
    return winner;
}

Token Board::get_Token(int row, int col) const
{
    return m_board[row][col];
}

Position *Board::get_valid_moves(int &num_moves)
{
    // Copy the board to the device
    int board_size = Board::BOARD_SIZE;

    // Allocate memory for valid moves on the device
    Position *device_valid_moves;
    hipMalloc(&device_valid_moves, board_size * board_size * sizeof(Position));

    // Initialize valid_moves_count on the host and copy to the device
    int valid_moves_count = 0;
    int *device_valid_moves_count;
    hipMalloc(&device_valid_moves_count, sizeof(int));
    hipMemcpy(device_valid_moves_count, &valid_moves_count, sizeof(int), hipMemcpyHostToDevice);

    dim3 block(8, 8);
    dim3 grid(BOARD_SIZE / block.x + 1, BOARD_SIZE / block.y + 1);
    valid_moves_kernel<<<grid, block>>>(d_board, board_size, device_valid_moves, device_valid_moves_count);

    // Copy the result back to the host
    hipMemcpy(&valid_moves_count, device_valid_moves_count, sizeof(int), hipMemcpyDeviceToHost);
    Position *host_valid_moves = new Position[valid_moves_count];
    hipMemcpy(host_valid_moves, device_valid_moves, valid_moves_count * sizeof(Position), hipMemcpyDeviceToHost);
    num_moves = valid_moves_count;
    // Free device memory
    // clear_space();
    hipFree(device_valid_moves);
    hipFree(device_valid_moves_count);

    return host_valid_moves;
}
bool Board::is_draw() const
{
    int num_moves = 0;
    Position *valid_moves = get_valid_moves(num_moves);
    delete[] valid_moves;
    return num_moves == 0;
}