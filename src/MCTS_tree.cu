#include "hip/hip_runtime.h"
#include "../include/MCTS_tree.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <queue>

__global__ void simulatekernel(Node *children, long long rate)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // int stride = blockDim.x * gridDim.x;
    hiprandState_t state;
    Node *parent = &children[i];
    hiprand_init(587, i, 0, &state);
    long long int start = clock64();
    long long int end = start;
    double elapsedTime = static_cast<double>(end - start) / rate;
    while (elapsedTime < 1000)
    {
        if (!parent->expanded)
        {
            parent->expand();
        }
        // pick a random number between 0 and 1
        double random = hiprand_uniform(&state);

        int chosen = static_cast<int>(random * parent->num_children);
        Node *child = &parent->children[chosen];
        child->visited++;
        child->sims++;

        if (!child->expanded)
        {
            child->expand();
        }
        // Highly unoptimized - multiple calls to get-valid_moves
        if (child->board.has_winner())
        {

            Token won = child->board.get_winner();
            parent = child;
            while (parent != &children[i])
            {
                if (won == Token::BLACK)
                {
                    parent->score -= 5;
                }
                else
                {
                    parent->score += 5;
                }
                parent = parent->parent;
            }
            if (won == Token::BLACK)
            {
                parent->score -= 5;
            }
            else
            {
                parent->score += 5;
            }
        }
        else if (child->board.is_draw())
        {
            int player = child->player;
            parent = child;
            while (parent != &children[i])
            {
                if (player == 1)
                {
                    parent->score -= 2;
                }
                else
                {
                    parent->score += 2;
                }
                parent = parent->parent;
                player = parent->player;
            }
            if (player == 1)
            {
                parent->score -= 2;
            }
            else
            {
                parent->score += 2;
            }
        }

        end += clock64();
    }
}
MonteCarloTree::MonteCarloTree(Board board, int player, Position move)
{
    root = new Node();
    root->children = new Node[16 * 16];
    root->board = board;
    root->parent = nullptr;
    root->player = player;
    root->visited = 0;
    root->sims = 0;
    root->wins = 0;
    root->score = 0;
    root->move = move;
    // root->expand();
}

MonteCarloTree::~MonteCarloTree()
{
    delete root;
}

// void expand(Node *node)
// {

// }

void MonteCarloTree::print_tree()
{
    // print_node(root);
}
/*
void MonteCarloTree::print_node(Node *node)
{
    std::cout << "Move made - row: " << node->move.row << ", col: " << node->move.col << std::endl;
    for (int i = 0; i < sizeof(node->children) / sizeof(node->children[0]); i++)
    {
        print_node(node->children[i]);
    }
}

void MonteCarloTree::print_node(Node *node, int depth)
{
    if (depth < 0)
        return;
    std::cout << "Move made - row: " << node->move.row << ", col: " << node->move.col << std::endl;
    for (int i = 0; i < sizeof(node->children) / sizeof(node->children[0]); i++)
    {
        print_node(node->children[i], depth - 1);
    }
}
*/
void MonteCarloTree::set_root(Node *node)
{
    root = node;
}

Node *MonteCarloTree::get_parent(Node *node)
{
    /*
    std::queue<Node *> q;
    q.push(root);

    while (!q.empty())
    {
        Node *current = q.front();
        q.pop();
        for (Node *child : current->children)
        {
            if (child == node)
            {
                return current;
            }
            q.push(child);
        }
    }
    */
    return nullptr;
}

Position MonteCarloTree::simulate(Node *node)
{
    node->expand();
    Node *childs;
    hipMalloc(&childs, 16 * 16 * sizeof(Node));
    hipMemcpy(childs, &node->children, 16 * 16 * sizeof(Node), hipMemcpyHostToDevice);

    dim3 block(8, 8);
    dim3 grid(16 / block.x + 1, 16 / block.y + 1);

    int temp_rate;

    hipError_t cudaStat = hipDeviceGetAttribute(&temp_rate, hipDeviceAttributeClockRate, 0);

    long long rate = (long long)temp_rate;
    simulatekernel<<<grid, block>>>(childs, rate);

    hipMemcpy(node->children, &childs, 16 * 16 * sizeof(Node), hipMemcpyHostToDevice);
    hipFree(childs);
    Position move = node->children[0].move;
    int max_score = node->children[0].score;
    for (int i = 0; i < sizeof(node->children) / sizeof(node->children[0]); i++)
    {
        if (node->children[i].score > max_score)
        {
            max_score = node->children[i].score;
            move = node->children[i].move;
        }
    }

    return move;
    // Code to select best possibble action. If all equal should we randomize?
}